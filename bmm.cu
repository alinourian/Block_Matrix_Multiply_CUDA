#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// TILEX and TILEY are used to set the number of threads in a CUDA block

#define TILEX 32
#define TILEY 16

#define L (TILEX <= TILEY ? TILEX:TILEY)

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n/TILEX,n/TILEY);
	return dimGrid;
}

dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX,TILEY);
	return dimBlock;
}

__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {

	int i = by * TILEY + ty;
	int j = bx * TILEX + tx;
	
	float sum = 0;
	
	__shared__ float as[TILEY][4 * L];
	__shared__ float bs[4 * L][TILEX];	

	for (int p = 0; p < (n/L); p += 4){

		if(tx < TILEY){
			as[ty][tx] = ad[((i)<<(m)) + (L*p+tx)];
			as[ty][tx + L] = ad[((i)<<(m)) + (L*(p+1)+tx)];
			as[ty][tx + 2*L] = ad[((i)<<(m)) + (L*(p+2)+tx)];
			as[ty][tx + 3*L] = ad[((i)<<(m)) + (L*(p+3)+tx)];
		}
		if (ty < TILEX){
			bs[ty][tx] = bd[((L*p + ty)<<(m)) + j];
			bs[ty + L][tx] = bd[((L * (p+1) + ty) << (m)) + j];
			bs[ty + 2*L][tx] = bd[((L * (p+2) + ty) << (m)) + j];
			bs[ty + 3*L][tx] = bd[((L * (p+3) + ty) << (m)) + j];
		}
		__syncthreads();
		
		for (int k = 0; k < 4 * L; k++){ 
			sum += as[ty][k] * bs[k][tx];
		}
		__syncthreads();
	}
	
	cd [((i) << (m)) + j] = sum;
}